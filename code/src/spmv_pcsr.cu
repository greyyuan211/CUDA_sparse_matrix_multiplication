#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "mmio.h"

#define threadsPerBlock 64
#define sizeSharedMemory 1024
#define BlockDim 1024
#define ITER 3

template <typename T>
__global__ void spmv_pcsr_kernel1(T * d_val,T * d_vector,int * d_cols,int d_nnz, T * d_v)
{
    	int tid = blockIdx.x * blockDim.x + threadIdx.x;
    	int icr = blockDim.x * gridDim.x;
    	while (tid < d_nnz){
		d_v[tid] = d_val[tid] * d_vector[d_cols[tid]];
        	tid += icr;
    	}
}

template <typename T>
__global__ void spmv_pcsr_kernel2(T * d_v,int * d_ptr,int N,T * d_out)
{
    	int gid = blockIdx.x * blockDim.x + threadIdx.x;
    	int tid = threadIdx.x;
    
    	__shared__ volatile int ptr_s[threadsPerBlock + 1];
    	__shared__ volatile T v_s[sizeSharedMemory];
 
   	// Load ptr into the shared memory ptr_s
    	ptr_s[tid] = d_ptr[gid];

	// Assign thread 0 of every block to store the pointer for the last row handled by the block into the last shared memory location
    	if (tid == 0) { 
    		if (gid + threadsPerBlock > N) {
	    		ptr_s[threadsPerBlock] = d_ptr[N];}
		else {
    	    		ptr_s[threadsPerBlock] = d_ptr[gid + threadsPerBlock];}
    	}
    	__syncthreads();

    	int temp = (ptr_s[threadsPerBlock] - ptr_s[0])/threadsPerBlock + 1;
    	int nlen = min(temp * threadsPerBlock,sizeSharedMemory);
    	T sum = 0;
    	int maxlen = ptr_s[threadsPerBlock];     
    	for (int i = ptr_s[0]; i < maxlen; i += nlen){
    		int index = i + tid;
    		__syncthreads();
    		// Load d_v into the shared memory v_s
    		for (int j = 0; j < nlen/threadsPerBlock;j++){
	    		if (index < maxlen) {
	        		v_s[tid + j * threadsPerBlock] = d_v[index];
	        		index += threadsPerBlock;
            		}
    		}
   	 	__syncthreads();

    		// Sum up the elements for a row
		if (!(ptr_s[tid+1] <= i || ptr_s[tid] > i + nlen - 1)) {
	   		int row_s = max(ptr_s[tid] - i, 0);
	    		int row_e = min(ptr_s[tid+1] -i, nlen);
	    		for (int j = row_s;j < row_e;j++){
				sum += v_s[j];
	    		}
		}	
    	}	
	// Write result
    	d_out[gid] = sum;
}

template <typename T>
void spmv_pcsr(MatrixInfo<T> * mat,T *vector,T *out) 
{
    	T *d_vector,*d_val, *d_out,*d_v;
    	int *d_cols, *d_ptr;
    	float time_taken;
    	double gflop = 2 * (double) mat->nz / 1e9;
    	float milliseconds = 0;
    	hipEvent_t start, stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);

	// Allocate memory on device
    	hipMalloc(&d_vector,mat->N*sizeof(T));
    	hipMalloc(&d_val,mat->nz*sizeof(T));
    	hipMalloc(&d_v,mat->nz*sizeof(T));
    	hipMalloc(&d_out,mat->M*sizeof(T));
    	hipMalloc(&d_cols,mat->nz*sizeof(int));
    	hipMalloc(&d_ptr,(mat->M+1)*sizeof(int));

	// Copy from host memory to device memory
    	hipMemcpy(d_vector,vector,mat->N*sizeof(T),hipMemcpyHostToDevice);
    	hipMemcpy(d_val,mat->val,mat->nz*sizeof(T),hipMemcpyHostToDevice);
    	hipMemcpy(d_cols,mat->cIndex,mat->nz*sizeof(int),hipMemcpyHostToDevice);
    	hipMemcpy(d_ptr,mat->rIndex,(mat->M+1)*sizeof(int),hipMemcpyHostToDevice);
    	hipMemset(d_out, 0, mat->M*sizeof(T));

	// Run the kernels and time them
    	hipEventRecord(start);
	for (int i = 0; i < ITER; i++) {
    		spmv_pcsr_kernel1<T><<<ceil(mat->nz/(float)BlockDim),BlockDim>>>(d_val,d_vector,d_cols,mat->nz,d_v);
    		spmv_pcsr_kernel2<T><<<ceil(mat->M/(float)threadsPerBlock),threadsPerBlock>>>(d_v,d_ptr,mat->M,d_out);
    	}
	hipEventRecord(stop);
    	hipEventSynchronize(stop);
    	hipEventElapsedTime(&milliseconds, start, stop);
   
	// Copy from device memory to host memory
    	hipMemcpy(out, d_out, mat->M*sizeof(T), hipMemcpyDeviceToHost);

	// Free device memory
    	hipFree(d_vector);
    	hipFree(d_val);
    	hipFree(d_cols);
    	hipFree(d_ptr); 
    	hipFree(d_out);
    	hipFree(d_v);
  	
	// Calculate and print out GFLOPs and GB/s
	double gbs = ((mat->N * sizeof(T)) + (mat->nz*sizeof(T) * 3) + (mat->M*sizeof(int)) + (mat->nz*sizeof(int)) + (mat->M*sizeof(T))) / (milliseconds/ITER) / 1e6; 
    	time_taken = (milliseconds/ITER)/1000.0; 
    	printf("Average time taken for %s is %f\n", "SpMV by GPU PCSR Algorithm",time_taken);
    	printf("Average GFLOP/s is %lf\n",gflop/time_taken);
	printf("Average GB/s is %lf\n\n",gbs);
}
