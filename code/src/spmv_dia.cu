#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "mmio.h"

#define BlockDim 1024
#define ITER 3


__global__ void spmv_dia_kernel ( const int num_rows ,const int num_cols , const int num_diags , const int * offsets , 
                                    const float * data , const float * x , float * y)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if( row < num_rows ){
        float dot = 0;
        for ( int n = 0; n < num_diags ; n ++){
            int col = row + offsets [n ];
            float val = data [ num_rows * n + row ];
            if( col >= 0 && col < num_cols )
                dot += val * x [ col ];
        }
        y[ row ] += dot ;
    }
}


