#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "mmio.h"

#define BlockDim 1024
#define ITER 3


__global__ void
spmv_ell_kernel ( const int num_rows , const int num_cols , const int num_cols_per_row , const int * indices , const float * data , const float * x , float * y)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x ;
    if( row < num_rows ){
        float dot = 0;
        for ( int n = 0; n < num_cols_per_row ; n ++){
            int col = indices [ num_rows * n + row ];
            float val = data [ num_rows * n + row ];
            if( val != 0)
                dot += val * x [ col ];
        }
        y[ row ] += dot ;
    }
}